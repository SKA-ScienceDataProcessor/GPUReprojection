
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <math.h>
#include <stdlib.h>

#define PRJERR_NULL_POINTER 1
#define PRJERR_BAD_PIX_SET_SIN 2
#define PRJERR_BAD_WORLD_SET_SIN 3

#define SIN 1

#define SIZEX 343
#define SIZEY 100
#define SIZE (SIZEX*SIZEY)

#define XLL 0.45
#define XUR 1.44
#define YLL -0.98
#define YUR 0.01

#define R2D (180/3.1415926)
#define D2R (3.1415926/180)

#define PAD_SIZE 2
#define IMG_SIZE 4096
#define IMG_PAD (IMG_SIZE+2*PAD_SIZE)
#define IMGX0 0.1
#define IMGX1 0.3
#define IMGY0 -0.3
#define IMGY1 0.1


void checkCudaError(int line, const char* filename) {
   hipError_t err = hipGetLastError();
   if (err) std::cerr << "Cuda error " << err << "(" << hipGetErrorString(err) <<
                         ") on line " << line << " of file " << filename << std::endl;
}
struct prjprm {
   int flag;
   double pv[3];
   double x0, y0;
   double w[4]; 
   double r0;
   int bounds;
};

__device__
void sinx2s_dev(double xi, double eta, double xoff, double yoff, double scale, 
                double aoff, double boff, double coff, int nx, int ny, 
                int sxy, int spt, double *x, double *y, double *phi, 
                double *theta, int *stat);
__device__
void sinx2s_dev(double xi, double eta, double xoff, double yoff, double scale, 
                double aoff, double boff, double coff, int nx, int ny, 
                int sxy, int spt, double *x, double *y, double *phi, 
                double *theta, int *stat)
{
  int mx, my, status;
  const double tol = 1.0e-13;
  double a, b, c, d, r2, sinth1, sinth2, sinthe, x0, x1, xy, y0, y02,
         y1, z;
  int ix, iy;


  //TODO What's this?
  if (ny > 0) {
    mx = nx;
    my = ny;
  } else {
    mx = 1;
    my = 1;
    ny = nx;
  }

  status = 0;


  /* Do x dependence. */

  /* Do y dependence. */
  iy=threadIdx.y + blockIdx.y * blockDim.y;
  ix=threadIdx.x + blockIdx.x * blockDim.x;
  if (ix>mx || iy>my) return;
    y0 = (y[sxy*iy] + yoff)*scale;
    y02 = y0*y0;

      /* Compute intermediaries. */
      x0 = (x[sxy*ix] + xoff)*scale;
      r2 = x0*x0 + y02;

      if (boff == 0.0) {
        /* Orthographic projection. */
        if (r2 != 0.0) {
          phi[ix*spt+mx*iy*spt] = atan2(x0, -y0);
        } else {
          phi[ix*spt+mx*iy*spt] = 0.0;
        }

        if (r2 < 0.5) {
          theta[ix*spt+mx*iy*spt] = acos(sqrt(r2));
        } else if (r2 <= 1.0) {
          theta[ix*spt+mx*iy*spt] = asin(sqrt(1.0 - r2));
        } else {
          stat[ix*spt+mx*iy*spt] = 1;
          if (!status) status = PRJERR_BAD_PIX_SET_SIN;
          return;
        }

      } else {
        /* "Synthesis" projection. */
        xy = x0*xi + y0*eta;

        if (r2 < 1.0e-10) {
          /* Use small angle formula. */
          z = r2/2.0;
          theta[ix*spt+mx*iy*spt] = 90.0 - R2D*sqrt(r2/(1.0 + xy));

        } else {
          a = aoff;
          b = xy + boff;
          c = r2 - xy - xy + coff;
          d = b*b - a*c;

          /* Check for a solution. */
          if (d < 0.0) {
            phi[ix*spt+mx*iy*spt] = 0.0;
            theta[ix*spt+mx*iy*spt] = 0.0;
            stat[ix*spt+mx*iy*spt] = 1;
            if (!status) status = PRJERR_BAD_PIX_SET_SIN;
            return;
          }
          d = sqrt(d);

          /* Choose solution closest to pole. */
          sinth1 = (-b + d)/a;
          sinth2 = (-b - d)/a;
          sinthe = (sinth1 > sinth2) ? sinth1 : sinth2;
          if (sinthe > 1.0) {
            if (sinthe-1.0 < tol) {
              sinthe = 1.0;
            } else {
              sinthe = (sinth1 < sinth2) ? sinth1 : sinth2;
            }
          }

          if (sinthe < -1.0) {
            if (sinthe+1.0 > -tol) {
              sinthe = -1.0;
            }
          }

          if (sinthe > 1.0 || sinthe < -1.0) {
            phi[ix*spt+mx*iy*spt] = 0.0;
            theta[ix*spt+mx*iy*spt] = 0.0;
            stat[ix*spt+mx*iy*spt] = 1;
            if (!status) status = PRJERR_BAD_PIX_SET_SIN;
            return;
          }

          theta[ix*spt+mx*iy*spt] = 0.0;
           asin(sinthe);
          z = 1.0 - sinthe;
        }

        x1 = -y0 + eta*z;
        y1 =  x0 -  xi*z;
        if (x1 == 0.0 && y1 == 0.0) {
          phi[ix*spt+mx*iy*spt] = 0.0;
        } else {
          phi[ix*spt+mx*iy*spt] = atan2(y1,x1);
        }
      }
   return;
}
//__device__
int sinx2s_alt(struct prjprm *prj, int nx, int ny, int sxy, int spt, double *x, double *y, double *phi, double *theta, int *stat)
{
  int mx, my, status;
  const double tol = 1.0e-13;
  double a, b, c, d, eta, r2, sinth1, sinth2, sinthe, x0, xi, x1, xy, y0, y02,
         y1, z;
  int ix, iy;


  /* Initialize. */
  if (prj == 0x0) return PRJERR_NULL_POINTER;
  if (prj->flag != SIN) {
    return 4;
  }

  xi  = prj->pv[1];
  eta = prj->pv[2];

  if (ny > 0) {
    mx = nx;
    my = ny;
  } else {
    mx = 1;
    my = 1;
    ny = nx;
  }

  status = 0;


  /* Do y dependence. */
  for (iy = 0; iy < ny; iy++) {
    y0 = (y[sxy*iy] + prj->y0)*prj->w[0];
    y02 = y0*y0;

    for (ix = 0; ix < mx; ix++) {
      
      x0 = (x[sxy*ix] + prj->x0)*prj->w[0];
      r2 = x0*x0 + y02;

      if (prj->w[1] == 0.0) {
        /* Orthographic projection. */
        if (r2 != 0.0) {
          phi[ix*spt+mx*iy*spt] = atan2(x0, -y0);
        } else {
          phi[ix*spt+mx*iy*spt] = 0.0;
        }

        if (r2 < 0.5) {
          theta[ix*spt+mx*iy*spt] = acos(sqrt(r2));
        } else if (r2 <= 1.0) {
          theta[ix*spt+mx*iy*spt] = asin(sqrt(1.0 - r2));
        } else {
          stat[ix*spt+mx*iy*spt] = 1;
          if (!status) status = PRJERR_BAD_PIX_SET_SIN;
          continue;
        }

      } else {
        /* "Synthesis" projection. */
        xy = x0*xi + y0*eta;

        if (r2 < 1.0e-10) {
          /* Use small angle formula. */
          z = r2/2.0;
          theta[ix*spt+mx*iy*spt] = 90.0 - R2D*sqrt(r2/(1.0 + xy));

        } else {
          a = prj->w[2];
          b = xy - prj->w[1];
          c = r2 - xy - xy + prj->w[3];
          d = b*b - a*c;

          /* Check for a solution. */
          if (d < 0.0) {
            phi[ix*spt+mx*iy*spt] = 0.0;
            theta[ix*spt+mx*iy*spt] = 0.0;
            stat[ix*spt+mx*iy*spt] = 1;
            if (!status) status = PRJERR_BAD_PIX_SET_SIN;
            continue;
          }
          d = sqrt(d);

          /* Choose solution closest to pole. */
          sinth1 = (-b + d)/a;
          sinth2 = (-b - d)/a;
          sinthe = (sinth1 > sinth2) ? sinth1 : sinth2;
          if (sinthe > 1.0) {
            if (sinthe-1.0 < tol) {
              sinthe = 1.0;
            } else {
              sinthe = (sinth1 < sinth2) ? sinth1 : sinth2;
            }
          }

          if (sinthe < -1.0) {
            if (sinthe+1.0 > -tol) {
              sinthe = -1.0;
            }
          }

          if (sinthe > 1.0 || sinthe < -1.0) {
            phi[ix*spt+mx*iy*spt] = 0.0;
            theta[ix*spt+mx*iy*spt] = 0.0;
            stat[ix*spt+mx*iy*spt] = 1;
            if (!status) status = PRJERR_BAD_PIX_SET_SIN;
            continue;
          }

          theta[ix*spt+mx*iy*spt] = asin(sinthe);
          z = 1.0 - sinthe;
        }

        x1 = -y0 + eta*z;
        y1 =  x0 -  xi*z;
        if (x1 == 0.0 && y1 == 0.0) {
          phi[ix*spt+mx*iy*spt] = 0.0;
        } else {
          phi[ix*spt+mx*iy*spt] = atan2(y1,x1);
        }
      }

      stat[ix*spt+mx*iy*spt] = 0;
    }
  }

  return 0;
}
//__device__
int sinx2s(struct prjprm *prj, int nx, int ny, int sxy, int spt, double *x, double *y, double *phi, double *theta, int *stat)
{
  int mx, my, rowlen, rowoff, status;
  const double tol = 1.0e-13;
  double a, b, c, d, eta, r2, sinth1, sinth2, sinthe, x0, xi, x1, xy, y0, y02,
         y1, z;
  int ix, iy, *statp;
  const double *xp, *yp;
  double *phip, *thetap;


  /* Initialize. */
  if (prj == 0x0) return PRJERR_NULL_POINTER;
  if (prj->flag != SIN) {
    return 4;
  }

  xi  = prj->pv[1];
  eta = prj->pv[2];

  if (ny > 0) {
    mx = nx;
    my = ny;
  } else {
    mx = 1;
    my = 1;
    ny = nx;
  }

  status = 0;


  /* Do x dependence. */
  xp = x;
  rowoff = 0;
  rowlen = nx*spt;
  for (ix = 0; ix < nx; ix++, rowoff += spt, xp += sxy) {
    x0 = (*xp + prj->x0)*prj->w[0];

    phip = phi + rowoff;
    for (iy = 0; iy < my; iy++) {
      *phip = x0;
      phip += rowlen;
    }
  }


  /* Do y dependence. */
  yp = y;
  phip   = phi;
  thetap = theta;
  statp  = stat;
  for (iy = 0; iy < ny; iy++, yp += sxy) {
    y0 = (*yp + prj->y0)*prj->w[0];
    y02 = y0*y0;

    for (ix = 0; ix < mx; ix++, phip += spt, thetap += spt) {
      /* Compute intermediaries. */
      x0 = *phip;
      r2 = x0*x0 + y02;

      if (prj->w[1] == 0.0) {
        /* Orthographic projection. */
        if (r2 != 0.0) {
          *phip = atan2(x0, -y0);
        } else {
          *phip = 0.0;
        }

        if (r2 < 0.5) {
          *thetap = acos(sqrt(r2));
        } else if (r2 <= 1.0) {
          *thetap = asin(sqrt(1.0 - r2));
        } else {
          *(statp++) = 1;
          if (!status) status = PRJERR_BAD_PIX_SET_SIN;
          continue;
        }

      } else {
        /* "Synthesis" projection. */
        xy = x0*xi + y0*eta;

        if (r2 < 1.0e-10) {
          /* Use small angle formula. */
          z = r2/2.0;
          *thetap = 90.0 - R2D*sqrt(r2/(1.0 + xy));

        } else {
          a = prj->w[2];
          b = xy - prj->w[1];
          c = r2 - xy - xy + prj->w[3];
          d = b*b - a*c;

          /* Check for a solution. */
          if (d < 0.0) {
            *phip = 0.0;
            *thetap = 0.0;
            *(statp++) = 1;
            if (!status) status = PRJERR_BAD_PIX_SET_SIN;
            continue;
          }
          d = sqrt(d);

          /* Choose solution closest to pole. */
          sinth1 = (-b + d)/a;
          sinth2 = (-b - d)/a;
          sinthe = (sinth1 > sinth2) ? sinth1 : sinth2;
          if (sinthe > 1.0) {
            if (sinthe-1.0 < tol) {
              sinthe = 1.0;
            } else {
              sinthe = (sinth1 < sinth2) ? sinth1 : sinth2;
            }
          }

          if (sinthe < -1.0) {
            if (sinthe+1.0 > -tol) {
              sinthe = -1.0;
            }
          }

          if (sinthe > 1.0 || sinthe < -1.0) {
            *phip = 0.0;
            *thetap = 0.0;
            *(statp++) = 1;
            if (!status) status = PRJERR_BAD_PIX_SET_SIN;
            continue;
          }

          *thetap = asin(sinthe);
          z = 1.0 - sinthe;
        }

        x1 = -y0 + eta*z;
        y1 =  x0 -  xi*z;
        if (x1 == 0.0 && y1 == 0.0) {
          *phip = 0.0;
        } else {
          *phip = atan2(y1,x1);
        }
      }

      *(statp++) = 0;
    }
  }

  return 0;
}

/*--------------------------------------------------------------------------*/
__device__
void sins2x_dev(double r0, double scale, double x0, double y0, double sintheta0,
               double costheta0, int bounds, int nphi, int ntheta, int spt, 
               int sxy, double *phi, double *theta, double *x, double *y, int *stat);

__device__
void sins2x_dev(double r0, double scale, double x0, double y0, double sintheta0,
               double costheta0, int bounds, int nphi, int ntheta, int spt, 
               int sxy, double *phi, double *theta, double *x, double *y, int *stat)

{
  int mphi, mtheta, status;
  double cosphi, costhe, sinphi, r, t, z, z1, z2;
  register int iphi, itheta, istat;


  if (ntheta > 0) {
    mphi   = nphi;
    mtheta = ntheta;
  } else {
    mphi   = 1;
    mtheta = 1;
    ntheta = nphi;
  }

  status = 0;

  /* Do theta dependence. */
  itheta = threadIdx.y + blockIdx.y * blockDim.y;
  iphi = threadIdx.x + blockIdx.x * blockDim.x;
  if (iphi>mphi || itheta>mtheta) return;
  //for (itheta = 0; itheta < ntheta; itheta++) {
    t = (90.0 - fabs(theta[itheta*spt]))*D2R;
    if (t < 1.0e-5) {
      if (theta[itheta*spt] > 0.0) {
         z = t*t/2.0;
      } else {
         z = 2.0 - t*t/2.0;
      }
      costhe = t;
    } else {
      z = 1.0 - sin(theta[itheta*spt]);
      costhe = cos(theta[itheta*spt]);
    }
    r = r0*costhe;

    if (scale == 0.0) {
      /* Orthographic projection. */
      istat = 0;
      if (bounds&1) {
        if (theta[itheta*spt] < 0.0) {
          istat = 1;
          if (!status) status = PRJERR_BAD_WORLD_SET_SIN;
        }
      }

      //for (iphi = 0; iphi < mphi; iphi++, xp += sxy, yp += sxy) {
        sincos(phi[iphi*sxy], &sinphi, &cosphi); 
        x[iphi*sxy+mphi*sxy*itheta] =  r*sinphi - x0;
        y[iphi*sxy+mphi*sxy*itheta] =  -r*cosphi - y0;
        stat[iphi*sxy+mphi*sxy*itheta] = istat;
      //}

    } else {
      /* "Synthesis" projection. */
      z *= r0;
      z1 = sintheta0*z - x0;
      z2 = costheta0*z - y0;

      //for (iphi = 0; iphi < mphi; iphi++) {
        istat = 0;
        sincos(phi[iphi*sxy], &sinphi, &cosphi); 
        if (bounds&1) {
          t = -atan(sintheta0*(sinphi) - costheta0*(cosphi));
          if (theta[itheta*spt] < t) {
            istat = 1;
            if (!status) status = PRJERR_BAD_WORLD_SET_SIN;
          }
        }

        x[iphi*sxy+mphi*sxy*itheta] = r*sinphi + z1;
        y[iphi*sxy+mphi*sxy*itheta] = -r*cosphi + z2;
        stat[iphi*sxy+mphi*sxy*itheta] = istat;
      //}
    }
  //}

  //return status;
}
int sins2x_alt(prjprm *prj, int nphi, int ntheta, int spt, int sxy, double *phi, double *theta, double *x, double *y, int *stat)

{
  int mphi, mtheta, status;
  double cosphi, costhe, sinphi, r, t, z, z1, z2;
  register int iphi, itheta, istat;


  if (ntheta > 0) {
    mphi   = nphi;
    mtheta = ntheta;
  } else {
    mphi   = 1;
    mtheta = 1;
    ntheta = nphi;
  }

  status = 0;

  /* Do theta dependence. */
  for (itheta = 0; itheta < ntheta; itheta++) {
    t = (90.0 - fabs(theta[itheta*spt]))*D2R;
    if (t < 1.0e-5) {
      if (theta[itheta*spt] > 0.0) {
         z = t*t/2.0;
      } else {
         z = 2.0 - t*t/2.0;
      }
      costhe = t;
    } else {
      z = 1.0 - sin(theta[itheta*spt]);
      costhe = cos(theta[itheta*spt]);
    }
    r = prj->r0*costhe;

    if (prj->w[1] == 0.0) {
      /* Orthographic projection. */
      istat = 0;
      if (prj->bounds&1) {
        if (theta[itheta*spt] < 0.0) {
          istat = 1;
          if (!status) status = PRJERR_BAD_WORLD_SET_SIN;
        }
      }

      for (iphi = 0; iphi < mphi; iphi++) {
        sincos(phi[iphi*sxy], &sinphi, &cosphi); 
        x[iphi*sxy+mphi*sxy*itheta] =  r*sinphi - prj->x0;
        y[iphi*sxy+mphi*sxy*itheta] =  -r*cosphi - prj->y0;
        stat[iphi*sxy+mphi*sxy*itheta] = istat;
      }

    } else {
      /* "Synthesis" projection. */
      z *= prj->r0;
      z1 = prj->pv[1]*z - prj->x0;
      z2 = prj->pv[2]*z - prj->y0;

      for (iphi = 0; iphi < mphi; iphi++) {
        istat = 0;
        sincos(phi[iphi*sxy], &sinphi, &cosphi); 
        if (prj->bounds&1) {
          t = -atan(prj->pv[1]*(sinphi) - prj->pv[2]*(cosphi));
          if (theta[itheta*spt] < t) {
            istat = 1;
            if (!status) status = PRJERR_BAD_WORLD_SET_SIN;
          }
        }

        x[iphi*sxy+mphi*sxy*itheta] = r*sinphi + z1;
        y[iphi*sxy+mphi*sxy*itheta] = -r*cosphi + z2;
        stat[iphi*sxy+mphi*sxy*itheta] = istat;
      }
    }
  }

  return status;
}
int sins2x(prjprm *prj, int nphi, int ntheta, int spt, int sxy, double *phi, double *theta, double *x, double *y, int *stat)

{
  int mphi, mtheta, rowlen, rowoff, status;
  double cosphi, costhe, sinphi, r, t, z, z1, z2;
  register int iphi, itheta, istat, *statp;
  register const double *phip, *thetap;
  register double *xp, *yp;


  if (ntheta > 0) {
    mphi   = nphi;
    mtheta = ntheta;
  } else {
    mphi   = 1;
    mtheta = 1;
    ntheta = nphi;
  }

  status = 0;


  /* Do phi dependence. */
  phip = phi;
  rowoff = 0;
  rowlen = nphi*sxy;
  for (iphi = 0; iphi < nphi; iphi++, rowoff += sxy, phip += spt) {
    sincos(*phip, &sinphi, &cosphi);

    xp = x + rowoff;
    yp = y + rowoff;
    for (itheta = 0; itheta < mtheta; itheta++) {
      *xp = sinphi;
      *yp = cosphi;
      xp += rowlen;
      yp += rowlen;
    }
  }


  /* Do theta dependence. */
  thetap = theta;
  xp = x;
  yp = y;
  statp = stat;
  for (itheta = 0; itheta < ntheta; itheta++, thetap += spt) {
    t = (90.0 - fabs(*thetap))*D2R;
    if (t < 1.0e-5) {
      if (*thetap > 0.0) {
         z = t*t/2.0;
      } else {
         z = 2.0 - t*t/2.0;
      }
      costhe = t;
    } else {
      z = 1.0 - sin(*thetap);
      costhe = cos(*thetap);
    }
    r = prj->r0*costhe;

    if (prj->w[1] == 0.0) {
      /* Orthographic projection. */
      istat = 0;
      if (prj->bounds&1) {
        if (*thetap < 0.0) {
          istat = 1;
          if (!status) status = PRJERR_BAD_WORLD_SET_SIN;
        }
      }

      for (iphi = 0; iphi < mphi; iphi++, xp += sxy, yp += sxy) {
        *xp =  r*(*xp) - prj->x0;
        *yp = -r*(*yp) - prj->y0;
        *(statp++) = istat;
      }

    } else {
      /* "Synthesis" projection. */
      z *= prj->r0;
      z1 = prj->pv[1]*z - prj->x0;
      z2 = prj->pv[2]*z - prj->y0;

      for (iphi = 0; iphi < mphi; iphi++, xp += sxy, yp += sxy) {
        istat = 0;
        if (prj->bounds&1) {
          t = -atan(prj->pv[1]*(*xp) - prj->pv[2]*(*yp));
          if (*thetap < t) {
            istat = 1;
            if (!status) status = PRJERR_BAD_WORLD_SET_SIN;
          }
        }

        *xp =  r*(*xp) + z1;
        *yp = -r*(*yp) + z2;
        *(statp++) = istat;
      }
    }
  }

  return status;
}
__global__ void sinx2s_kernel(double xi, double eta, double xoff, double yoff, double scale,
                double aoff, double boff, double coff, int nx, int ny, int sxy, 
                int spt, double *x, double *y, double *phi, double *theta, 
                int *stat)                                                 {
  sinx2s_dev(xi, eta, xoff, yoff, scale, aoff, boff, coff, nx, ny, sxy, spt, 
             x, y, phi, theta, stat);
}
__global__ void sins2x_kernel(double r0, double scale, double x0, double y0, double sintheta0,
               double costheta0, int bounds, int nphi, int ntheta, int spt, 
               int sxy, double *phi, double *theta, double *x, double *y, int *stat) {
  sins2x_dev(r0, scale, x0, y0, sintheta0, costheta0, bounds, nphi, ntheta, sxy, spt, 
             phi, theta, x, y, stat);
}
__device__ void interp_dev(const double* x3, const double* y3, double2* img_orig, int sz, 
                              double xgrid, double ygrid, double2* img_out) {
      int z = threadIdx.x + blockIdx.x * blockDim.x;
      z += blockDim.x*gridDim.x*(threadIdx.y + blockIdx.y*blockDim.y); 
      if (z>=sz) return;
      double thisx = x3[z]-IMGX0;
      double thisy = y3[z]-IMGY0;
      int x0 = floorf(thisx/xgrid)+PAD_SIZE;
      double xfrac = thisx/xgrid-x0+PAD_SIZE;
      int y0 = floorf(thisy/ygrid)+PAD_SIZE;
      double yfrac = thisy/ygrid-y0+PAD_SIZE;
      int inx0 = IMG_PAD*y0+x0;
      inx0 %= IMG_PAD*IMG_PAD;
      double out_x = img_orig[inx0].x;
      double out_y = img_orig[inx0].y;
      out_x *= (1-xfrac)*(1-yfrac);
      out_y *= (1-xfrac)*(1-yfrac);
      out_x += (1-xfrac)*yfrac*img_orig[inx0+IMG_PAD].x;
      out_y += (1-xfrac)*yfrac*img_orig[inx0+IMG_PAD].y;
      out_x += xfrac*(1-yfrac)*img_orig[inx0+1].x;
      out_y += xfrac*(1-yfrac)*img_orig[inx0+1].y;
      out_x += xfrac*yfrac*img_orig[inx0+IMG_PAD+1].x;
      out_y += xfrac*yfrac*img_orig[inx0+IMG_PAD+1].y;
      img_out[z].x = out_x;
      img_out[z].y = out_y;
      //img_out[z].x = g00.x*(1-xfrac)*(1-yfrac)+g01.x*(1-xfrac)*yfrac+g10.x*xfrac*(1-yfrac)+g11.x*xfrac*yfrac;
      //img_out[z].y = g00.y*(1-xfrac)*(1-yfrac)+g01.y*(1-xfrac)*yfrac+g10.y*xfrac*(1-yfrac)+g11.y*xfrac*yfrac;
     
}
__global__ void interp_kernel(const double* x3, const double* y3, double2* img_orig, int sz, 
                              double xgrid, double ygrid, double2* img_out) {
  interp_dev(x3, y3, img_orig, sz, xgrid, ygrid, img_out);
}
__global__ void coord_convert(double xi, double eta, double xoff, double yoff, double scale_in,
               double aoff, double boff, double coff, int nx, int ny, 
               double r0, double scale_out, double x0, double y0, double sintheta0,
               double costheta0, int bounds, int nphi, int ntheta, 
               int sxy, int spt, double *x, double *y, double *phi, double *theta, 
               double2* img_orig, int sz, double xgrid, double ygrid, double2* img_out, int *stat) {
  sinx2s_dev(xi, eta, xoff, yoff, scale_in, aoff, boff, coff, nx, ny, sxy, spt, 
             x, y, phi, theta, stat);
  sins2x_dev(r0, scale_out, x0, y0, sintheta0, costheta0, bounds, nphi, ntheta, sxy, spt, 
             phi, theta, x, y, stat);
  interp_dev(x, y, img_orig, sz, xgrid, ygrid, img_out);
}


int main(void) {
   double *x, *y, *phi, *theta, *x2, *y2;

   int *stat;

   struct prjprm prj;

   x = (double*)malloc(sizeof(double)*SIZE);
   y = (double*)malloc(sizeof(double)*SIZE);
   x2 = (double*)malloc(sizeof(double)*SIZE);
   y2 = (double*)malloc(sizeof(double)*SIZE);
   phi = (double*)malloc(sizeof(double)*SIZE);
   theta = (double*)malloc(sizeof(double)*SIZE);
   stat = (int*)malloc(sizeof(int)*SIZE);

   /***   Initialize ***/
   srand(2541617);
   prj.flag = SIN;
   prj.pv[0] = (rand()*1.0)/RAND_MAX;
   prj.pv[1] = (rand()*1.0)/RAND_MAX;
   prj.pv[2] = (rand()*1.0)/RAND_MAX;
   prj.x0=0;
   prj.y0=0;
   prj.r0=0.663;
   prj.w[0] = (rand()*1.0)/RAND_MAX;
   prj.w[1] = 0.0;
   //prj.w[1] = (rand()*1.0)/RAND_MAX;
   prj.w[2] = (rand()*1.0)/RAND_MAX;
   prj.w[3] = (rand()*1.0)/RAND_MAX;
   prj.bounds = 0;

   for (int z=0;z<SIZE;z++) {
      x[z] = XLL + (z%SIZEX)*(XUR/SIZEX);
      y[z] = YLL + (z/SIZEX)*(YUR/SIZEY);
   }

   double2* img_orig;
   double2* img_out;
   double2* img_out2;
   img_orig = (double2*)malloc(sizeof(double2)*IMG_PAD*IMG_PAD);
   img_out = (double2*)malloc(sizeof(double2)*IMG_PAD*IMG_PAD);
   img_out2 = (double2*)malloc(sizeof(double2)*IMG_PAD*IMG_PAD);
   if (!img_orig || !img_out || !img_out2) std::cerr << "ERROR. Failed CPU alloc." <<std::endl;
   for (int z=0;z<IMG_PAD*IMG_PAD;z++) {
      img_orig[z].x = (rand()*1.0)/RAND_MAX;
      img_orig[z].y = (rand()*1.0)/RAND_MAX;
   }

   /*** Execute x2s two ways ***/
   if (sinx2s(&prj, SIZEX, SIZEY, 1, 1, x, y, phi, theta, stat)) 
                        std::cout << "ERROR in sinx2s" << std::endl;

   /*** Execute s2x two ways ***/
   if (sins2x(&prj, SIZEX, SIZEY, 1, 1, phi, theta, x2, y2, stat)) 
                        std::cout << "ERROR in sins2x" << std::endl;

   double xmin, xmax, ymin, ymax;
   xmin = ymin = 100000000;
   xmax = ymax = -100000000;
   for (int z=0;z<SIZE;z++) {
      if (x2[z]<xmin) xmin = x2[z]; 
      if (y2[z]<ymin) ymin = y2[z]; 
      if (x2[z]>xmax) xmax = x2[z]; 
      if (y2[z]>ymax) ymax = y2[z]; 
   }
   std::cout << xmin << ", " << ymin << " -- " << xmax << ", " << ymax << std::endl;
   double xgrid = (IMGX1-IMGX0)/IMG_SIZE;
   double ygrid = (IMGY1-IMGY0)/IMG_SIZE;
   for (int z=0;z<SIZE;z++) {
      double thisx = x2[z]-IMGX0;
      double thisy = y2[z]-IMGY0;
      int x0 = floorf(thisx/xgrid)+PAD_SIZE;
      double xfrac = thisx/xgrid-x0+PAD_SIZE;
      int y0 = floorf(thisy/ygrid)+PAD_SIZE;
      double yfrac = thisy/ygrid-y0+PAD_SIZE;
      int inx0 = IMG_PAD*y0+x0;
      double2 g00 = img_orig[inx0];
      double2 g01 = img_orig[inx0+IMG_PAD];
      double2 g10 = img_orig[inx0+1];
      double2 g11 = img_orig[inx0+IMG_PAD+1];
      img_out[z].x = g00.x*(1-xfrac)*(1-yfrac)+g01.x*(1-xfrac)*yfrac+g10.x*xfrac*(1-yfrac)+g11.x*xfrac*yfrac;
      img_out[z].y = g00.y*(1-xfrac)*(1-yfrac)+g01.y*(1-xfrac)*yfrac+g10.y*xfrac*(1-yfrac)+g11.y*xfrac*yfrac;
   }

   /*** GPU memory ***/
   double2 *dimg_orig, *dimg_out;
   hipMalloc(&dimg_orig, sizeof(double2)*IMG_PAD*IMG_PAD);
   hipMalloc(&dimg_out, sizeof(double2)*IMG_PAD*IMG_PAD);
   if (!dimg_orig || !dimg_out) std::cerr << "ERROR: Failed GPU allocation." << std::endl;
   hipMemcpy(dimg_orig, img_orig, sizeof(double2)*IMG_PAD*IMG_PAD, hipMemcpyHostToDevice);
   checkCudaError(__LINE__,__FILE__);

   double *dx, *dy, *dphi, *dtheta;
   int *dstat;
   hipMalloc(&dx, sizeof(double)*SIZE);
   hipMalloc(&dy, sizeof(double)*SIZE);
   hipMalloc(&dphi, sizeof(double)*SIZE);
   hipMalloc(&dtheta, sizeof(double)*SIZE);
   hipMalloc(&dstat, sizeof(int)*SIZE);
   
   hipMemcpy(dx, x, sizeof(double)*SIZE, hipMemcpyHostToDevice);
   hipMemcpy(dy, y, sizeof(double)*SIZE, hipMemcpyHostToDevice);
   checkCudaError(__LINE__,__FILE__);

   /*** Compute on GPU ***/
   coord_convert<<<dim3(1,SIZEY),SIZEX>>>(prj.pv[1], prj.pv[2], prj.x0, prj.y0, prj.w[0], 
                     prj.w[2], -prj.w[1], prj.w[3], SIZEX, SIZEY, prj.r0, prj.w[1], 
                     prj.x0, prj.y0, prj.pv[1], prj.pv[2], prj.bounds, SIZEX, SIZEY, 
                     1, 1, dx, dy, dphi, dtheta, dimg_orig, IMG_SIZE*IMG_SIZE, xgrid, ygrid,
                     dimg_out, dstat);
   checkCudaError(__LINE__,__FILE__);
   hipMemcpy(img_out2, dimg_out, sizeof(double2)*IMG_PAD*IMG_PAD, hipMemcpyDeviceToHost);
   checkCudaError(__LINE__,__FILE__);

   std::cout << "Check results against CPU..." << std::endl;

   for (int z=0;z<SIZE;z+=1000) {
      if (fabs(img_out2[z].x-img_out[z].x) > 0.00001 ||
          fabs(img_out2[z].y-img_out[z].y) > 0.00001  ) {
         std::cout << "Mismatch for z = " << z << ": " << img_out2[z].x << ", " <<img_out2[z].y << " != "
                   << img_out[z].x << ", " << img_out[z].y << std::endl;
      }
   }
   free(x);
   free(y);
   free(x2);
   free(y2);
   free(phi);
   free(theta);
   free(stat);
   free(img_orig);
   free(img_out);
   free(img_out2);

   hipFree(dx);
   hipFree(dy);
   hipFree(dphi);
   hipFree(dtheta);
   hipFree(dstat);
   hipFree(dimg_orig);
   hipFree(dimg_out);

   
}
